
#include <hip/hip_runtime.h>
#include <atomic>
#include <cmath>
#include <fstream>
#include <functional>
#include <iomanip>
#include <limits>
#include <mutex>
#include <stdexcept>
#include <string>
#include <thread>
#include <vector>

namespace param {
const int n_steps = 200000;
const double dt = 60;
const double eps = 1e-3;
const double G = 6.674e-11;
__device__ double gravity_device_mass(double m0, double t) {
    return m0 + 0.5 * m0 * fabs(sin(t / 6000));
}
const double planet_radius = 1e7;
const double missile_speed = 1e6;
double get_missile_cost(double t) { return 1e5 + 1e3 * t; }

const double eps2 = eps * eps;
const double planet_radius2 = planet_radius * planet_radius;
}  // namespace param

#define BlockSize 32

struct Vec3 {
    double x, y, z;
};

struct Prop {
    double m;
    bool is_device;
};

void read_input(const char* filename, int& n, int& planet, int& asteroid,
    std::vector<Vec3>& q, std::vector<Vec3>& v, std::vector<Prop>& prop) {
    std::ifstream fin(filename);
    fin >> n >> planet >> asteroid;
    q.resize(n);
    v.resize(n);
    prop.resize(n);
    for (int i = 0; i < n; i++) {
        std::string type;
        fin >> q[i].x >> q[i].y >> q[i].z >> v[i].x >> v[i].y >> v[i].z >> prop[i].m >>
            type;
        prop[i].is_device = type == "device";
    }
}

void write_output(const char* filename, double min_dist, int hit_time_step,
    int gravity_device_id, double missile_cost) {
    std::ofstream fout(filename);
    fout << std::scientific
         << std::setprecision(std::numeric_limits<double>::digits10 + 1) << min_dist
         << '\n'
         << hit_time_step << '\n'
         << gravity_device_id << ' ' << missile_cost << '\n';
}

template <bool with_devices>
__device__ void run_step(int step, int n, Vec3* q, Vec3* nq, Vec3* v, Prop* p) {
    int i = blockIdx.x;
    double ax = 0;
    double ay = 0;
    double az = 0;
    for (int j = threadIdx.x; j < n; j += blockDim.x) {
        if (i == j) continue;
        double mj = p[j].m;
        if (p[j].is_device) {
            mj = with_devices ? param::gravity_device_mass(mj, step * param::dt) : 0;
        }
        double dx = q[j].x - q[i].x;
        double dy = q[j].y - q[i].y;
        double dz = q[j].z - q[i].z;
        double dist3 = rsqrt(dx * dx + dy * dy + dz * dz + param::eps2);
        dist3 = dist3 * dist3 * dist3;
        ax += mj * dx * dist3;
        ay += mj * dy * dist3;
        az += mj * dz * dist3;
    }
    atomicAdd(&v[i].x, param::G * param::dt * ax);
    atomicAdd(&v[i].y, param::G * param::dt * ay);
    atomicAdd(&v[i].z, param::G * param::dt * az);
    __syncthreads();
    if (threadIdx.x == 0) {
        nq[i].x = q[i].x + v[i].x * param::dt;
        nq[i].y = q[i].y + v[i].y * param::dt;
        nq[i].z = q[i].z + v[i].z * param::dt;
    }
}

__global__ void prob1(double* answer, int planet, int asteroid, int step, int n, Vec3* q,
    Vec3* nq, Vec3* v, Prop* p) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        double dx = q[planet].x - q[asteroid].x;
        double dy = q[planet].y - q[asteroid].y;
        double dz = q[planet].z - q[asteroid].z;
        double distance = sqrt(dx * dx + dy * dy + dz * dz);
        *answer = fmin(*answer, distance);
    }
    run_step<false>(step + 1, n, q, nq, v, p);
}

__global__ void prob2(int* answer, int planet, int asteroid, int step, int n, Vec3* q,
    Vec3* nq, Vec3* v, Prop* p) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        double dx = q[planet].x - q[asteroid].x;
        double dy = q[planet].y - q[asteroid].y;
        double dz = q[planet].z - q[asteroid].z;
        double distance2 = dx * dx + dy * dy + dz * dz;
        if (distance2 < param::planet_radius2) {
            *answer = min(*answer, step);
        }
    }
    run_step<true>(step + 1, n, q, nq, v, p);
}

__global__ void prob3(int* reach_at_step, int device, int planet, int asteroid, int step,
    int n, Vec3* q, Vec3* nq, Vec3* v, Prop* p) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        double dx = q[planet].x - q[device].x;
        double dy = q[planet].y - q[device].y;
        double dz = q[planet].z - q[device].z;
        double distance2 = dx * dx + dy * dy + dz * dz;
        double traveled = step * param::dt * param::missile_speed;
        if (distance2 < traveled * traveled) {
            *reach_at_step = min(*reach_at_step, step);
            p[device].m = 0;
        }
        dx = q[planet].x - q[asteroid].x;
        dy = q[planet].y - q[asteroid].y;
        dz = q[planet].z - q[asteroid].z;
        if (dx * dx + dy * dy + dz * dz < param::planet_radius2) {
            *reach_at_step = -2;
        }
    }
    run_step<true>(step + 1, n, q, nq, v, p);
}

template <class T>
T* xCudaMallocVector(const std::vector<T>& host) {
    T* devPtr;
    hipMalloc(&devPtr, host.size() * sizeof(T));
    return devPtr;
}

template <class T>
void xCudaCopyVector(T* devPtr, const std::vector<T>& host) {
    hipMemcpy(devPtr, host.data(), host.size() * sizeof(T), hipMemcpyHostToDevice);
}

template <class T>
void xCudaReturnVector(T* devPtr, std::vector<T>& host) {
    hipMemcpy(host.data(), devPtr, host.size() * sizeof(T), hipMemcpyDeviceToHost);
}

int main(int argc, char** argv) {
    if (argc != 3) {
        throw std::runtime_error("must supply 2 arguments");
    }
    int n, planet, asteroid;
    std::vector<Vec3> q, v;
    std::vector<Prop> p;
    read_input(argv[1], n, planet, asteroid, q, v, p);

    std::vector<std::function<std::function<void()>(Vec3*, Vec3*, Vec3*, Prop*)>> tasks;

    // Problem 1
    double min_dist = std::numeric_limits<double>::infinity();
    tasks.push_back([&](Vec3* dq, Vec3* dnq, Vec3* dv,
                        Prop* dp) -> std::function<void()> {
        xCudaCopyVector(dq, q);
        xCudaCopyVector(dv, v);
        xCudaCopyVector(dp, p);
        double* dev_min_dist;
        hipMalloc(&dev_min_dist, sizeof min_dist);
        hipMemcpy(dev_min_dist, &min_dist, sizeof min_dist, hipMemcpyHostToDevice);
        for (int step = 0; step <= param::n_steps; step++) {
            prob1<<<n, BlockSize>>>(dev_min_dist, planet, asteroid, step, n, dq, dnq, dv, dp);
            std::swap(dq, dnq);
        }
        hipMemcpy(&min_dist, dev_min_dist, sizeof min_dist, hipMemcpyDeviceToHost);
        printf("prob1, cgle=%d\n", hipGetLastError());
        return []() {};
    });

    // Problem 2
    int hit_time_step = param::n_steps + 1;
    tasks.push_back(
        [&](Vec3* dq, Vec3* dnq, Vec3* dv, Prop* dp) -> std::function<void()> {
            int* dev_hit_time_step;
            xCudaCopyVector(dq, q);
            xCudaCopyVector(dv, v);
            xCudaCopyVector(dp, p);
            hipMalloc(&dev_hit_time_step, sizeof hit_time_step);
            hipMemcpy(dev_hit_time_step, &hit_time_step, sizeof hit_time_step,
                hipMemcpyHostToDevice);
            for (int step = 0; step <= param::n_steps; step++) {
                prob2<<<n, BlockSize>>>(
                    dev_hit_time_step, planet, asteroid, step, n, dq, dnq, dv, dp);
                std::swap(dq, dnq);
            }
            hipMemcpy(&hit_time_step, dev_hit_time_step, sizeof hit_time_step,
                hipMemcpyDeviceToHost);
            if (hit_time_step > param::n_steps) {
                hit_time_step = -2;
            }
            printf("prob2, cgle=%d\n", hipGetLastError());
            return []() {};
        });

    // Problem 3
    // TODO
    int gravity_device_id = -1;
    double missile_cost = param::n_steps + 1;
    for (int i = 0; i < n; i++) {
        if (p[i].is_device) {
            tasks.push_back(
                [&, i](Vec3* dq, Vec3* dnq, Vec3* dv, Prop* dp) -> std::function<void()> {
                    int host_reached_at = param::n_steps + 1;
                    int* dev_reached_at;
                    hipMalloc(&dev_reached_at, sizeof host_reached_at);
                    hipMemcpy(dev_reached_at, &host_reached_at, sizeof host_reached_at,
                        hipMemcpyHostToDevice);
                    xCudaCopyVector(dq, q);
                    xCudaCopyVector(dv, v);
                    xCudaCopyVector(dp, p);
                    for (int step = 0; step <= param::n_steps; step++) {
                        prob3<<<n, BlockSize>>>(dev_reached_at, i, planet, asteroid, step, n, dq,
                            dnq, dv, dp);
                        std::swap(dq, dnq);
                    }
                    hipMemcpy(&host_reached_at, dev_reached_at, sizeof host_reached_at,
                        hipMemcpyDeviceToHost);
                    printf("prob3, cgle=%d, %d, %d\n", hipGetLastError(), i,
                        host_reached_at);
                    if (host_reached_at != -2 and host_reached_at != param::n_steps + 1) {
                        return [&]() {
                            if (host_reached_at < missile_cost) {
                                missile_cost = host_reached_at;
                                gravity_device_id = i;
                            }
                        };
                    } else {
                        return []() {};
                    }
                });
        }
    }

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    std::atomic_int task(0);
    std::vector<std::thread> threads;
    std::mutex mux;
    for (int dev = 0; dev < deviceCount; dev++) {
        threads.emplace_back([&, dev]() {
            hipSetDevice(dev);
            Vec3* dq = xCudaMallocVector(q);
            Vec3* dnq = xCudaMallocVector(q);
            Vec3* dv = xCudaMallocVector(v);
            Prop* dp = xCudaMallocVector(p);
            for (int i = task.fetch_add(1); i < tasks.size(); i = task.fetch_add(1)) {
                auto done = tasks[i](dq, dnq, dv, dp);
                {
                    std::lock_guard<std::mutex> l(mux);
                    done();
                }
            }
        });
    }
    for (auto& thread : threads) {
        thread.join();
    }

    if (gravity_device_id == -1) {
        missile_cost = 0;
    } else {
        missile_cost = param::get_missile_cost(missile_cost * param::dt);
    }

    write_output(argv[2], min_dist, hit_time_step, gravity_device_id, missile_cost);
}
